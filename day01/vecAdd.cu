
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void add(int *a, int *b, int *c) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    c[index] = a[index] + b[index];
}

void random_ints(int *array, int size) {
    for (int i = 0; i < size; i++) {
        array[i] = rand() % 100;  // Random integers between 0 and 99
    }
}

#define N (2048*2048)
#define THREADS_PER_BLOCK 512

int main(void) {
    int *a, *b, *c;			// host copies of a, b, c
    int *d_a, *d_b, *d_c;		// device copies of a, b, c
    int size = N * sizeof(int);

    // Alloc space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Alloc space for host copies of a, b, c and setup input values
    a = (int *)malloc(size); random_ints(a, N);
    b = (int *)malloc(size); random_ints(b, N);
    c = (int *)malloc(size);

    // Copy inputs to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Launch add() kernel on GPU
    add<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_a, d_b, d_c);

    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Cleanup
    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    return 0;
}
